#include "hip/hip_runtime.h"
// #undef __SIZEOF_FLOAT128__
#include <iostream>
#include <system_error>

#include <rmm/mr/device/cuda_memory_resource.hpp>
#include <rmm/mr/device/device_memory_resource.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>
#include <rmm/mr/device/per_device_resource.hpp>
#include <rmm/device_uvector.hpp>
#include <thrust/uninitialized_fill.h>
#include <thrust/system/cuda/execution_policy.h>

#include "my_kernel.h"

using namespace std;
constexpr int con = 3;

__global__ void my_kernel() {
  printf("inside my kernel con is %d\n", con);
  // call the real kernel <<< >>>
}

void my_kernel_wrapper() {
  printf("calling kernel\n");
  my_kernel<<<1, 1>>>();
}


hipError_t example() {
  rmm::mr::cuda_memory_resource cuda_mr;
  // Construct a resource that uses a coalescing best-fit pool allocator
  rmm::mr::pool_memory_resource<rmm::mr::cuda_memory_resource> pool_mr{&cuda_mr};
  rmm::mr::set_current_device_resource(&pool_mr); // Updates the current device resource pointer to `pool_mr`
  rmm::mr::device_memory_resource * mr = rmm::mr::get_current_device_resource(); 
  rmm::cuda_stream_view s{};
 
  // Allocates *uninitialized* device memory on stream `s` sufficient for 100 ints using the
  // supplied resource `mr`
  rmm::device_uvector<int> uv(100, s, mr);
  // std::cout << uv.begin() << ":" << uv.end() << std::endl;
  thrust::uninitialized_fill(thrust::cuda::par.on(s.value()), uv.begin(), uv.end(), 0);
  return hipDeviceSynchronize();  							     
}